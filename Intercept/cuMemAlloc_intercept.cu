#include <dlfcn.h>
#include <string.h>
#include <iostream>
#include <hip/hip_runtime.h>

extern "C" {void *__libc_dlsym(void *map, const char *name);}
extern "C" {void *__libc_dlopen_mode(const char *name, int maddArgumentode);}

typedef void *(*fnDlsym)(void *, const char *);
static void *real_dlsym(void *handle, const char *symbol)
{
    static fnDlsym internal_dlsym = (fnDlsym)__libc_dlsym(__libc_dlopen_mode("libdl.so.2", RTLD_LAZY), "dlsym");
    return (*internal_dlsym)(handle, symbol);
}

static void *realFunctions;

hipError_t hipMalloc (hipDeviceptr_t* dptr, size_t bytesize)
{
    std::cout << "@@@@==hipMalloc hooked====" << std::endl;
    if (realFunctions == NULL) realFunctions = real_dlsym(RTLD_NEXT, "hipMalloc");
    return  ((hipError_t (*)(hipDeviceptr_t*, size_t))realFunctions)(dptr, bytesize);
}

void *dlsym(void *handle, const char *symbol)
{
    if (strcmp(symbol, "hipMalloc") == 0) {
        if(realFunctions == NULL) realFunctions = real_dlsym(handle, symbol);
        return (void*)(&hipMalloc);
    }

    return (real_dlsym(handle, symbol));
}
